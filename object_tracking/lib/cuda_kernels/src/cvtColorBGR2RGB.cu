#include "hip/hip_runtime.h"
#include "cuda_kernel.h"
#include <stdio.h>
#include <iostream>

#define BLOCK_SIZE 32

__global__ void cvtColorBGR2RGB_shared_kernel(unsigned char *d_rgb, unsigned char *d_bgr, int width)
{
    // +1 for padding due to bank conflict
    __shared__ unsigned char sharedMemory [BLOCK_SIZE][BLOCK_SIZE+1][3];

	// global index	
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	// local index
	int localIndexX = threadIdx.x;
	int localIndexY = threadIdx.y;
    // array index
    int index = indexY * width + indexX;

    // reading from global memory in coalesed manner in shared memory
    sharedMemory[localIndexX][localIndexY][0] = d_bgr[index*3];  // b
    sharedMemory[localIndexX][localIndexY][1] = d_bgr[index*3+1];  // g
    sharedMemory[localIndexX][localIndexY][2] = d_bgr[index*3+2];  // r

    // writing into global memory in coalesed fashion via data in shared memory
    d_rgb[index*3] = sharedMemory[localIndexX][localIndexY][2];   // r
    d_rgb[index*3+1] = sharedMemory[localIndexX][localIndexY][1]; // g
    d_rgb[index*3+2] = sharedMemory[localIndexX][localIndexY][0];   // b
}

void cvtColorBGR2RGB(unsigned char *d_rgb, unsigned char *d_bgr, int width, int height)
{
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 gridSize( (width+BLOCK_SIZE-1)/BLOCK_SIZE, (height+BLOCK_SIZE-1)/BLOCK_SIZE, 1);
    cvtColorBGR2RGB_shared_kernel<<<gridSize, blockSize>>>(d_rgb, d_bgr, width);   
    hipDeviceSynchronize();
}